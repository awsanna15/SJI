#include "hip/hip_runtime.h"

#include "ResizeImage.h"



#include <hip/hip_runtime.h>

#define PI 3.14159265359




// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
 //       TRACE("line(%i) : CUDA Runtime API error %d: %s.\n", line, (int)err, hipGetErrorString(err));
		 hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}


__global__ void
MirrorHorizontal_kernel( unsigned short *SrcDest, int wSrcDst, int hSrcDst, int step)
{
	int x= threadIdx.x + blockIdx.x*blockDim.x;
	int y= threadIdx.y + blockIdx.y*blockDim.y;

	if(x<=int((wSrcDst+1)/2) && y<hSrcDst)
	{
		int offset=x+y*step;
		int offset2=(wSrcDst-1-x)+y*step;
		unsigned short tmp=SrcDest[offset];
		SrcDest[offset]=SrcDest[offset2];
		SrcDest[offset2]=tmp;

	}
}


__global__ void
Rotate8u_kernel(unsigned char *dest, unsigned char *src, int wDst, int hDst, float cos_deg, float sin_deg)
{

	int ix = threadIdx.x + blockIdx.x*blockDim.x;
	int iy = threadIdx.y + blockIdx.y*blockDim.y;

	int resultPix = 0;
	int offset = ix + iy*wDst;

	bool bAvailable = false;

	if (ix<wDst && iy<hDst)
	{
		float xc = (float)wDst / 2;
		float yc = (float)hDst / 2;

		float x = ((float)ix - xc)*cos_deg - ((float)iy - yc)*sin_deg + xc;
		float y = ((float)ix - xc)*sin_deg + ((float)iy - yc)*cos_deg + yc;

		bAvailable = true;

		if (x >= 0.0f && x <= wDst - 1 && y >= 0 && y <= hDst - 1)
		{
			int i = int(x);
			int j = int(y);

			float ki1 = (x - (float)i);
			float ki0 = 1.0f - ki1;

			float kj1 = (y - (float)j);
			float kj0 = 1.0f - kj1;

			float SumPix = 0.0f;
			float SumK = 0.0f;


			SumPix += ki0*kj0*src[i + j*wDst];
			SumK += ki0*kj0;

			if (i + 1 <= wDst - 1)
			{
				SumPix += ki1*kj0*src[(i + 1) + j*wDst];
				SumK += ki1*kj0;
			}
			if (j + 1 <= hDst - 1)
			{
				SumPix += ki0*kj1*src[i + (j + 1)*wDst];
				SumK += ki0*kj1;
			}
			if (i + 1 <= wDst - 1 && j + 1 <= hDst - 1)
			{
				SumPix += ki1*kj1*src[(i + 1) + (j + 1)*wDst];
				SumK += ki1*kj1;
			}
			resultPix = max(0, min(32767, int(SumPix / SumK + 0.5)));

		}
		else
		{
			resultPix = 0;
		}

	}

	__syncthreads();

	if (bAvailable)
	{
		dest[offset] = resultPix;
	}
}

__global__ void
Threshold_kernel(unsigned short *dest, unsigned short *src, int wSrcDst, int hSrcDst, unsigned short threshold)
{
	int x= threadIdx.x + blockIdx.x*blockDim.x;
	int y= threadIdx.y + blockIdx.y*blockDim.y;

	if(x<wSrcDst && y<hSrcDst)
	{
		int offset=x+y*wSrcDst;
		unsigned short tmp=src[offset];
		if(tmp>0)
		{
			dest[offset]=tmp;
		}
		else
		{
			dest[offset]=0;
		}
	}
}

//********************************************************************************
__global__ void
LocalContrast_kernel(unsigned short *dest, unsigned short *src, int w, int h, float fVal1, float fVal2)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;

	if (x < w && y < h)
	{
		float sumAverage = (float)0.0f;
		unsigned short currentFrameSample1 = src[x + y*w];
		int cnt = 0;
		for (int i = -3; i <= 3; i++)
		{
			for (int j = -3; j <= 3; j++)
			{
				if ((x + i) >= 0 && (x + i) < w && (y + j) > 0 && (y + j) < h)
				{
					sumAverage += src[(x + i) + (y + j)*w];
					cnt++;
				}
				
			}
		}
		unsigned short currentFrameSample2 = sumAverage / cnt;

		if (currentFrameSample1<currentFrameSample2)
		{
			dest[x + y*w] = currentFrameSample2 + (currentFrameSample1 - currentFrameSample2)*fVal1;
		}
		else
		{
			dest[x + y*w] = currentFrameSample2 + (currentFrameSample1 - currentFrameSample2) / fVal2;
		}

	}
}
//********************************************************************************
__global__ void
RealLocalContrast_kernel(unsigned short *dest, unsigned short *src, int w, int h, float fVal1)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;

	if (x < w && y < h)
	{
		float sumAverage = (float)0.0f;
		unsigned short currentFrameSample1 = src[x + y*w];
		int cnt = 0;
		for (int i = -7; i <= 7; i++)
		{
			for (int j = -7; j <= 7; j++)
			{
				if ((x + i) >= 0 && (x + i) < w && (y + j) > 0 && (y + j) < h)
				{
					sumAverage += src[(x + i) + (y + j)*w];
					cnt++;
				}
				
			}
		}
		unsigned short currentFrameSample2 = sumAverage / cnt;
		float tmp=(float)currentFrameSample2 + float(currentFrameSample1 - currentFrameSample2)*fVal1;
		if(tmp<0)
		{
			tmp=0;
		}
		else if(tmp>32767)
		{
			tmp=32767;
		}
		dest[x + y*w] =  unsigned short(tmp);

	}
}

__global__ void
ShiftXYMapImage_kernel(unsigned short *dev_Dst, unsigned short *Image_dev, unsigned short *ResultImage_dev, int w, int h, float sinTheta, float cosTheta, int val)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;

	if (x < w && y < h && x >= 0 && y >= 0)
	{
		float dx = (float(ResultImage_dev[x + y*w])-128.0f)*cosTheta;
		float dy = (float(ResultImage_dev[x + y*w])-128.0f)*sinTheta;
		int xsrc = x + dx;
		int ysrc = y + dy;

		if (xsrc >= 0 && xsrc < w && ysrc >= 0 && ysrc < h)
		{
			dev_Dst[x + y*w] = Image_dev[xsrc + ysrc*w];
		}
		else
		{
			if (val >= 0)
			{
				dev_Dst[x + y*w] = val;
			}
			else
			{
				dev_Dst[x + y*w] = Image_dev[x + y*w];
			}

		}
	}
	else
	{
		if (val >= 0)
		{
			dev_Dst[x + y*w] = val;
		}
		else
		{
			dev_Dst[x + y*w] = Image_dev[x + y*w];
		}

	}
}

__global__ void
ShiftXYImage_kernel(unsigned short *dest, unsigned short *src, int w, int h, int dx, int dy, int val)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;

	if (x < w && y < h && x>=0 && y>=0)
	{
		int xsrc= x + dx;
		int ysrc= y + dy;
		
		if (xsrc >= 0 && xsrc < w && ysrc >= 0 && ysrc < h)
		{
			dest[x + y*w] = src[xsrc + ysrc*w];
		}
		else
		{
			if (val >= 0)
			{
				dest[x + y*w] = val;
			}
			else
			{
				dest[x + y*w] = src[x + y*w];
			}
			
		}
	}
	
}

//************************** ResizeDown *******************************************
__global__ void
ResizeDown_kernel(unsigned short *dest, unsigned short *src, int wSrc, int hSrc, int wDst, int hDst)
{
	int x= threadIdx.x + blockIdx.x*blockDim.x;
	int y= threadIdx.y + blockIdx.y*blockDim.y;

	if(x<wDst && y<hDst)
	{
		int offset=x+y*wDst;

		float aMin=(float)x/(float)wDst;
		float bMin=(float)y/(float)hDst;

		float aMax=(float)(x+1)/(float)wDst;
		float bMax=(float)(y+1)/(float)hDst;

		int a0=int(x*(float)wSrc/(float)wDst);
		int b0=int(y*(float)hSrc/(float)hDst);

		int xtmp=a0;
		int ytmp=b0;

		float SumPix=0.0f;
		float SumK=0.0f;
		float PixP=1.0f/(wSrc*hSrc);

		while((float)xtmp/wSrc < aMax)
		{
			ytmp=b0;
			while((float)ytmp/hSrc < bMax)
			{
				float fx0=max((float)xtmp/wSrc,aMin);
				float fy0=max((float)ytmp/hSrc,bMin);
				float fx1=min((float)(xtmp+1)/wSrc,aMax);
				float fy1=min((float)(ytmp+1)/hSrc,bMax);

				float k=(fx1-fx0)*(fy1-fy0)/PixP;

				if(xtmp<wSrc && ytmp<hSrc)
				{
					SumK+=k;
					SumPix+=k*src[xtmp+ytmp*wSrc]; 
				}
				ytmp++;
			}
			xtmp++;
		}
		if(SumPix>0)
		{
			dest[offset] = int(SumPix/SumK+0.5);
		}
		else
		{
			dest[offset] = 0;
		}

	
	}
   
}
//************************************************************************

//************************* ResizeUp **************************************
__global__ void
ResizeUp_kernel(unsigned short *dest, unsigned short *src, int wSrc, int hSrc, int wDst, int hDst)
{
	int x= threadIdx.x + blockIdx.x*blockDim.x;
	int y= threadIdx.y + blockIdx.y*blockDim.y;

	if(x<wDst && y<hDst)
	{
		int offset=x+y*wDst;

		float a0f=x*(float)wSrc/(float)wDst;
		float b0f=y*(float)hSrc/(float)hDst;
		int a0=int(a0f);
		int b0=int(b0f);

		float ki1=a0f-(float)a0;
		float ki0=1.0f-ki1;

		float kj1=b0f-(float)b0;
		float kj0=1.0f-kj1;

		float SumPix=0.0f;
		float SumK=0.0f;

	
		SumPix+=ki0*kj0*src[a0+b0*wSrc];
		SumK+=ki0*kj0;

		if(a0+1<wSrc)
		{
			SumPix+=ki1*kj0*src[(a0+1)+b0*wSrc];
			SumK+=ki1*kj0;
		}
		if(b0+1<hSrc)
		{
			SumPix+=ki0*kj1*src[a0+(b0+1)*wSrc];
			SumK+=ki0*kj1;
		}
		if(a0+1<wSrc && b0+1<hSrc)
		{
			SumPix+=ki1*kj1*src[(a0+1)+(b0+1)*wSrc];
			SumK+=ki1*kj1;
		}
		dest[offset] = int(SumPix/SumK+0.5);

	}

}



//************************** ResizeDown8u *******************************************
__global__ void
ResizeDown8u_kernel(unsigned char *dest, unsigned char *src, int wSrc, int hSrc, int wDst, int hDst)
{
	int x= threadIdx.x + blockIdx.x*blockDim.x;
	int y= threadIdx.y + blockIdx.y*blockDim.y;

	if(x<wDst && y<hDst)
	{
		int offset=x+y*wDst;

		float aMin=(float)x/(float)wDst;
		float bMin=(float)y/(float)hDst;

		float aMax=(float)(x+1)/(float)wDst;
		float bMax=(float)(y+1)/(float)hDst;

		int a0=int(x*(float)wSrc/(float)wDst);
		int b0=int(y*(float)hSrc/(float)hDst);

		int xtmp=a0;
		int ytmp=b0;

		float SumPix=0.0f;
		float SumK=0.0f;
		float PixP=1.0f/(wSrc*hSrc);

		while((float)xtmp/wSrc < aMax)
		{
			ytmp=b0;
			while((float)ytmp/hSrc < bMax)
			{
				float fx0=max((float)xtmp/wSrc,aMin);
				float fy0=max((float)ytmp/hSrc,bMin);
				float fx1=min((float)(xtmp+1)/wSrc,aMax);
				float fy1=min((float)(ytmp+1)/hSrc,bMax);

				float k=(fx1-fx0)*(fy1-fy0)/PixP;

				if(xtmp<wSrc && ytmp<hSrc)
				{
					SumK+=k;
					SumPix+=k*src[xtmp+ytmp*wSrc]; 
				}
				ytmp++;
			}
			xtmp++;
		}
		if(SumPix>0)
		{
			dest[offset] = char(SumPix/SumK+0.5);
		}
		else
		{
			dest[offset] = 0;
		}

	
	}
   
}
//************************************************************************

//************************* ResizeUp **************************************
__global__ void
ResizeUp8u_kernel(unsigned char *dest, unsigned char *src, int wSrc, int hSrc, int wDst, int hDst)
{
	int x= threadIdx.x + blockIdx.x*blockDim.x;
	int y= threadIdx.y + blockIdx.y*blockDim.y;

	if(x<wDst && y<hDst)
	{
		int offset=x+y*wDst;

		float a0f=x*(float)wSrc/(float)wDst;
		float b0f=y*(float)hSrc/(float)hDst;
		int a0=int(a0f);
		int b0=int(b0f);

		float ki1=a0f-(float)a0;
		float ki0=1.0f-ki1;

		float kj1=b0f-(float)b0;
		float kj0=1.0f-kj1;

		float SumPix=0.0f;
		float SumK=0.0f;

	
		SumPix+=ki0*kj0*src[a0+b0*wSrc];
		SumK+=ki0*kj0;

		if(a0+1<wSrc)
		{
			SumPix+=ki1*kj0*src[(a0+1)+b0*wSrc];
			SumK+=ki1*kj0;
		}
		if(b0+1<hSrc)
		{
			SumPix+=ki0*kj1*src[a0+(b0+1)*wSrc];
			SumK+=ki0*kj1;
		}
		if(a0+1<wSrc && b0+1<hSrc)
		{
			SumPix+=ki1*kj1*src[(a0+1)+(b0+1)*wSrc];
			SumK+=ki1*kj1;
		}
		dest[offset] = char(SumPix/SumK+0.5);

	}

}
__global__ void AddImages_kernel(unsigned short *Dest, unsigned short *SrcImage1, unsigned short *SrcImage2, int wDst, int hDst)
{
	int ix= threadIdx.x + blockIdx.x*blockDim.x;
	int iy= threadIdx.y + blockIdx.y*blockDim.y;
	int offset=ix+iy*wDst;

	if(ix<wDst && iy<hDst)
	{
		Dest[offset]=(SrcImage1[offset]+SrcImage2[offset])/2;
	}

}


__global__ void Set_kernel(unsigned short *SrcDest, int wDst, int hDst, unsigned short ValC)
{
	int x= threadIdx.x + blockIdx.x*blockDim.x;
	int y= threadIdx.y + blockIdx.y*blockDim.y;
	
	int offset=x+y*wDst;
	
	if(x<wDst && y<hDst)
	{
		SrcDest[offset] =ValC;
	}
   
}

__global__ void
CudaMemCopyRect_kernel(unsigned short *dev_result, unsigned short * dev_src, int iwSrc, int ihSrc, int iwDst, int ihDst, int sx, int sy)
{
	int xDst = threadIdx.x + blockIdx.x*blockDim.x;
	int yDst = threadIdx.y + blockIdx.y*blockDim.y;


	if (xDst < iwDst && yDst < ihDst && xDst>=0 && yDst>=0)
	{
		int xSrc = xDst + sx;
		int ySrc = yDst + sy;
		if (xSrc < iwSrc && ySrc < ihSrc && xSrc >= 0 && ySrc >= 0)
		{
			dev_result[xDst + yDst*iwDst] = dev_src[xSrc + ySrc*iwSrc];
		}
		else
		{ 
			dev_result[xDst + yDst*iwDst] = 0;
		}


	}

}

__global__ void 
ShiftImage_kernel(unsigned short *dest, const unsigned short *src, const int width, const int height, const float dx, const float dy)
{
	int xDst= threadIdx.x + blockIdx.x*blockDim.x;
	int yDst= threadIdx.y + blockIdx.y*blockDim.y;

	
	if(xDst<width && yDst<height)
	{
		int xSrc=xDst+int(dx>0 ? dx+0.5f:dx-0.5f);
		int ySrc=yDst+int(dy>0 ? dy+0.5f:dy-0.5f);

		dest[xDst+yDst*width]=src[xSrc+ySrc*width];
	}
}

__global__ void
Div32Image_kernel(unsigned short *dst, const int width, const int height, float *src, const int fwidth, const int fheight, float divc)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;


	if (x<width && y<height)
	{
	

		dst[x + y*width] = unsigned short((src[x + y*fwidth]/ divc)+0.5);
	}
}

__global__ void
ShiftAddImage_kernel(float *dest, const int fwidth, const int fheight, const unsigned short *src, const int width, const int height, const float dx, const float dy)
{
	int xDst = threadIdx.x + blockIdx.x*blockDim.x;
	int yDst = threadIdx.y + blockIdx.y*blockDim.y;


	if (xDst<width && yDst<height)
	{
		int xSrc = xDst + int(dx>0 ? dx + 0.5f : dx - 0.5f);
		int ySrc = yDst + int(dy>0 ? dy + 0.5f : dy - 0.5f);

		dest[xDst + yDst*fwidth] += float(src[xSrc + ySrc*width]);
	}
}

__global__ void
Rotate_kernel(unsigned short *dest, unsigned short *src, int wDst, int hDst, float cos_deg, float sin_deg)
{

	int ix= threadIdx.x + blockIdx.x*blockDim.x;
	int iy= threadIdx.y + blockIdx.y*blockDim.y;
	
	int resultPix=0;
	int offset=ix+iy*wDst;

	bool bAvailable=false;
	
	if(ix<wDst && iy<hDst && ix>=0 && iy>=0)
	{
		float xc = (float)wDst/2;
		float yc = (float)hDst/2;

		float x = ((float)ix-xc)*cos_deg - ((float)iy-yc)*sin_deg + xc;
		float y = ((float)ix-xc)*sin_deg + ((float)iy-yc)*cos_deg + yc;

		bAvailable=true;

		if(x>=0.0f && x<=wDst-1 && y>=0 && y<=hDst-1)
		{
			int i= int(x);
			int j= int(y);

			float ki1=(x-(float)i);
			float ki0=1.0f-ki1;

			float kj1=(y-(float)j);
			float kj0=1.0f-kj1;

			float SumPix=0.0f;
			float SumK=0.0f;

	
			SumPix+=ki0*kj0*src[i+j*wDst];
			SumK+=ki0*kj0;

			if(i+1<=wDst-1)
			{
				SumPix+=ki1*kj0*src[(i+1)+j*wDst];
				SumK+=ki1*kj0;
			}
			if(j+1<=hDst-1)
			{
				SumPix+=ki0*kj1*src[i+(j+1)*wDst];
				SumK+=ki0*kj1;
			}
			if(i+1<=wDst-1 && j+1<=hDst-1)
			{
				SumPix+=ki1*kj1*src[(i+1)+(j+1)*wDst];
				SumK+=ki1*kj1;
			}
			resultPix = max(0,min(32767,int(SumPix/SumK+0.5)));
			
		}
		else
		{
			resultPix = src[offset];
		}

	}

	__syncthreads();

	if(bAvailable)
	{
		dest[offset] =resultPix;
	}
}


__global__ void
Zoom_kernel2(unsigned short *dest, unsigned short *src, int wDst, int hDst, float fZoom,  int Val)
{
	int ix= threadIdx.x + blockIdx.x*blockDim.x;
	int iy= threadIdx.y + blockIdx.y*blockDim.y;
	
	int resultPix=0;
	int offset=ix+iy*wDst;

	bool bAvailable=false;
	
	if(ix<wDst && iy<hDst)
	{
		float x=((float)ix-(float)wDst/2)*fZoom;
		float y=((float)iy-(float)hDst/2)*fZoom;

		x+=(float)wDst/2;
		y+=(float)hDst/2;

		

		if(x>=0.0f && x<=wDst-1 && y>=0 && y<=hDst-1)
		{
			int i= int(x);
			int j= int(y);

			float ki1=(x-(float)i);
			float ki0=1.0f-ki1;

			float kj1=(y-(float)j);
			float kj0=1.0f-kj1;

			float SumPix=0.0f;
			float SumK=0.0f;

	        if(i>=0 && i<wDst && j>=0 && j<hDst)
            {
			    SumPix+=ki0*kj0*src[i+j*wDst];
			    SumK+=ki0*kj0;
            }

			if((i+1)>=0 && (i+1)<wDst && j>=0 && j<hDst)
			{
				SumPix+=ki1*kj0*src[(i+1)+j*wDst];
				SumK+=ki1*kj0;
			}

			if(i>=0 && i<wDst && (j+1)>=0 && (j+1)<hDst)
			{
				SumPix+=ki0*kj1*src[i+(j+1)*wDst];
				SumK+=ki0*kj1;
			}

			if((i+1)>=0 && (i+1)<wDst && (j+1)>=0 && (j+1)<hDst)
			{
				SumPix+=ki1*kj1*src[(i+1)+(j+1)*wDst];
				SumK+=ki1*kj1;
			}

			resultPix = max(0,min(32767,int(SumPix/SumK+0.5)));
			bAvailable = true;
		}
		else
		{
			bAvailable = true;
			if (Val >= 0)
			{
				resultPix = Val;
			}
			else
			{
				resultPix = src[ix + iy*wDst];
			}
			
		}

	}

	__syncthreads();

	if(bAvailable)
	{
		dest[offset] =resultPix;
	}
}


__global__ void
Zoom_kernel(unsigned short *dest, unsigned short *src, int wDst, int hDst, float fZoom, float dx, float dy)
{
	int ix= threadIdx.x + blockIdx.x*blockDim.x;
	int iy= threadIdx.y + blockIdx.y*blockDim.y;
	
	int resultPix=0;
	int offset=ix+iy*wDst;

	bool bAvailable=false;
	
	if(ix<wDst && iy<hDst)
	{
		float x=((float)ix-(float)wDst/2)*fZoom+dx;
		float y=((float)iy-(float)hDst/2)*fZoom+dy;

		x+=(float)wDst/2;
		y+=(float)hDst/2;

		bAvailable=true;

		if(x>=0.0f && x<=wDst-1 && y>=0 && y<=hDst-1)
		{
			int i= int(x);
			int j= int(y);

			float ki1=(x-(float)i);
			float ki0=1.0f-ki1;

			float kj1=(y-(float)j);
			float kj0=1.0f-kj1;

			float SumPix=0.0f;
			float SumK=0.0f;

	
			SumPix+=ki0*kj0*src[i+j*wDst];
			SumK+=ki0*kj0;

			if(i+1<=wDst-1)
			{
				SumPix+=ki1*kj0*src[(i+1)+j*wDst];
				SumK+=ki1*kj0;
			}
			if(j+1<=hDst-1)
			{
				SumPix+=ki0*kj1*src[i+(j+1)*wDst];
				SumK+=ki0*kj1;
			}
			if(i+1<=wDst-1 && j+1<=hDst-1)
			{
				SumPix+=ki1*kj1*src[(i+1)+(j+1)*wDst];
				SumK+=ki1*kj1;
			}
			resultPix = max(0,min(32767,int(SumPix/SumK+0.5)));
			
		}
		else
		{
			resultPix = 0;
		}

	}

	__syncthreads();

	if(bAvailable)
	{
		dest[offset] =resultPix;
	}
}


//************************************************************************

// Wrapper for the __global__ call that sets up the texture and threads
extern "C" void Zoom_Dev(unsigned short *dev_result, unsigned short *dev_src, int iwDst, int ihDst, float fZoom, float fx, float fy)
{
	dim3 blocks((iwDst+31)/32,(ihDst+15)/16);
	dim3 threads(32,16);

	Zoom_kernel<<<blocks, threads>>>(dev_result, dev_src, iwDst,  ihDst, fZoom, fx, fy);
}


extern "C" void AddImages_Dev(unsigned short *dev_result, unsigned short *dev_src1, unsigned short *dev_src2, int iwDst, int ihDst)
{
	dim3 blocks((iwDst+31)/32,(ihDst+15)/16);
	dim3 threads(32,16);

	AddImages_kernel<<<blocks, threads>>>(dev_result, dev_src1, dev_src2, iwDst, ihDst);
}


extern "C" void Rotate_Dev(unsigned short *dev_result, unsigned short *dev_src, int iwDst, int ihDst, float fAngleDeg)
{
	dim3 blocks((iwDst+31)/32,(ihDst+15)/16);
	dim3 threads(32,16);

	float cos_deg=cos(fAngleDeg*PI/180);
	float sin_deg=sin(fAngleDeg*PI/180);

	Rotate_kernel<<<blocks, threads>>>(dev_result, dev_src, iwDst,  ihDst, cos_deg, sin_deg);
}



extern "C" void ResizeImage_Dev(unsigned short *dev_result, unsigned short *dev_src, int iwSrc, int ihSrc, int iwDst, int ihDst)
{
	dim3 blocks((iwDst+31)/32,(ihDst+15)/16);
	dim3 threads(32,16);

	if(iwDst<iwSrc)
	{
		ResizeDown_kernel<<<blocks, threads>>>(dev_result, dev_src, iwSrc, ihSrc,  iwDst,  ihDst);
	}
	else
	{
		ResizeUp_kernel<<<blocks, threads>>>(dev_result, dev_src, iwSrc, ihSrc,  iwDst,  ihDst);
	}
}

extern "C" void Threshold_Dev(unsigned short *dest, unsigned short *src, int wSrcDst, int hSrcDst, unsigned short threshold)
{
	dim3 blocks((wSrcDst+31)/32,(hSrcDst+15)/16);
	dim3 threads(32,16);

	Threshold_kernel<<<blocks, threads>>>(dest, src, wSrcDst, hSrcDst, threshold);
}


extern "C" void MirrorHorizontal_dev(unsigned short *srcDev, int iwSrc, int ihSrc, int step)
{
	dim3 blocks((iwSrc+31)/32,(ihSrc+15)/16);
	dim3 threads(32,16);

	MirrorHorizontal_kernel<<<blocks, threads>>>( srcDev, iwSrc, ihSrc, step);
}

extern "C" void MirrorHorizontal(unsigned short *src, int iwSrc, int ihSrc, int step)
{
	unsigned short *dev_srcDst=NULL;

	checkCudaErrors(hipMalloc(&dev_srcDst, sizeof(unsigned short)*iwSrc*ihSrc));
    checkCudaErrors(hipMemcpy(dev_srcDst, src,sizeof(unsigned short)*iwSrc*ihSrc, hipMemcpyHostToDevice));

	MirrorHorizontal_dev(dev_srcDst, iwSrc, ihSrc, step);

	checkCudaErrors(hipMemcpy(src, dev_srcDst, iwSrc*ihSrc*sizeof(unsigned short), hipMemcpyDeviceToHost));


	checkCudaErrors(hipFree(dev_srcDst));
}

extern "C" void LocalContrast_dev(unsigned short *dev_srcDst, int iw, int ih, float fVal1, float fVal2)
{
	unsigned short *dev_tmp = NULL;
	(hipMalloc((void **)&dev_tmp, sizeof(unsigned short)*iw*ih));

	dim3 blocks((iw + 31) / 32, (ih + 15) / 16);
	dim3 threads(32, 16);


	LocalContrast_kernel << <blocks, threads >> >(dev_tmp, dev_srcDst, iw, ih, fVal1, fVal2);
	(hipMemcpy(dev_srcDst, dev_tmp, iw*ih * sizeof(unsigned short), hipMemcpyDeviceToDevice));
	(hipFree(dev_tmp));
}

extern "C" void LocalContrast(unsigned short *SrcDst, int iw, int ih, float fVal1, float fVal2)
{
	
	unsigned short *dev_srcDst = NULL;

	(hipMalloc(&dev_srcDst, sizeof(unsigned short)*iw*ih));
	(hipMemcpy(dev_srcDst, SrcDst, sizeof(unsigned short)*iw*ih, hipMemcpyHostToDevice));

	LocalContrast_dev(dev_srcDst, iw, ih, fVal1, fVal2);

	(hipMemcpy(SrcDst, dev_srcDst, iw*ih * sizeof(unsigned short), hipMemcpyDeviceToHost));
	
	(hipFree(dev_srcDst));
}

extern "C" void RealLocalContrast_dev(unsigned short *dev_srcDst, int iw, int ih, float fVal1)
{
	unsigned short *dev_tmp = NULL;
	(hipMalloc((void **)&dev_tmp, sizeof(unsigned short)*iw*ih));

	dim3 blocks((iw + 31) / 32, (ih + 15) / 16);
	dim3 threads(32, 16);


	RealLocalContrast_kernel << <blocks, threads >> >(dev_tmp, dev_srcDst, iw, ih, fVal1);
	(hipMemcpy(dev_srcDst, dev_tmp, iw*ih * sizeof(unsigned short), hipMemcpyDeviceToDevice));
	(hipFree(dev_tmp));
}

extern "C" void RealLocalContrast(unsigned short *SrcDst, int iw, int ih, float fVal1)
{
	
	unsigned short *dev_srcDst = NULL;

	(hipMalloc(&dev_srcDst, sizeof(unsigned short)*iw*ih));
	(hipMemcpy(dev_srcDst, SrcDst, sizeof(unsigned short)*iw*ih, hipMemcpyHostToDevice));

	RealLocalContrast_dev(dev_srcDst, iw, ih, fVal1);

	(hipMemcpy(SrcDst, dev_srcDst, iw*ih * sizeof(unsigned short), hipMemcpyDeviceToHost));
	
	(hipFree(dev_srcDst));
}

extern "C" void ShiftXYImage_dev(unsigned short *Dst_dev, unsigned short *Src_dev, int iw, int ih, int dx, int dy, int val)
{


	dim3 blocks((iw + 31) / 32, (ih + 15) / 16);
	dim3 threads(32, 16);


	ShiftXYImage_kernel << <blocks, threads >> >(Dst_dev, Src_dev, iw, ih, dx, dy, val);


}

extern "C" void ShiftXYZoomImage(unsigned short *SrcDst, int iw, int ih, int dx, int dy, float fZoom, int val)
{
	unsigned short *dev_tmp = NULL;
	unsigned short *dev_srcDst = NULL;



	(hipMalloc(&dev_srcDst, sizeof(unsigned short)*iw*ih));
	(hipMemcpy(dev_srcDst, SrcDst, sizeof(unsigned short)*iw*ih, hipMemcpyHostToDevice));

	(hipMalloc((void **)&dev_tmp, sizeof(unsigned short)*iw*ih));

	dim3 blocks((iw + 31) / 32, (ih + 15) / 16);
	dim3 threads(32, 16);


	ShiftXYImage_kernel << <blocks, threads >> >(dev_tmp, dev_srcDst, iw, ih, dx, dy, val);
	Zoom_kernel2 << <blocks, threads >> >(dev_srcDst, dev_tmp, iw, ih, fZoom, val);

	
	(hipMemcpy(SrcDst, dev_srcDst, iw*ih * sizeof(unsigned short), hipMemcpyDeviceToHost));


	(hipFree(dev_tmp));
	(hipFree(dev_srcDst));
}


extern "C" void ShiftXYMapImage_dev(unsigned short *dev_Dst, unsigned short *Image_dev, unsigned short *ResultImage_dev, int iw, int ih, float sinTheta, float cosTheta, int val)
{
	dim3 blocks((iw + 31) / 32, (ih + 15) / 16);
	dim3 threads(32, 16);

	ShiftXYMapImage_kernel << <blocks, threads >> > (dev_Dst, Image_dev, ResultImage_dev, iw, ih, sinTheta, cosTheta, val);
}

extern "C" void ShiftXYImage(unsigned short *SrcDst, int iw, int ih, int dx, int dy)
{
	unsigned short *dev_tmp = NULL;
	unsigned short *dev_srcDst = NULL;



	(hipMalloc(&dev_srcDst, sizeof(unsigned short)*iw*ih));
	(hipMemcpy(dev_srcDst, SrcDst, sizeof(unsigned short)*iw*ih, hipMemcpyHostToDevice));

	(hipMalloc((void **)&dev_tmp, sizeof(unsigned short)*iw*ih));

	dim3 blocks((iw + 31) / 32, (ih + 15) / 16);
	dim3 threads(32, 16);


	ShiftXYImage_kernel << <blocks, threads >> >(dev_tmp, dev_srcDst, iw, ih, dx, dy, -1);
	
	(hipMemcpy(SrcDst, dev_tmp, iw*ih * sizeof(unsigned short), hipMemcpyDeviceToHost));


	(hipFree(dev_tmp));
	(hipFree(dev_srcDst));
}

extern "C" void ResizeImage(unsigned short *dst, unsigned short *src, int iwSrc, int ihSrc, int iwDst, int ihDst)
{
	unsigned short *dev_result=NULL;
	unsigned short *dev_src=NULL;

	hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned short>();
	
	checkCudaErrors(hipMalloc(&dev_src, sizeof(unsigned short)*iwSrc*ihSrc));
    checkCudaErrors(hipMemcpy(dev_src, src,sizeof(unsigned short)*iwSrc*ihSrc, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc( (void **)&dev_result, iwDst*ihDst*sizeof(unsigned short)));

	dim3 blocks((iwDst+31)/32,(ihDst+15)/16);
	dim3 threads(32,16);

	if(iwDst<iwSrc)
	{
		ResizeDown_kernel<<<blocks, threads>>>(dev_result, dev_src, iwSrc, ihSrc,  iwDst,  ihDst);
	}
	else
	{
		ResizeUp_kernel<<<blocks, threads>>>(dev_result, dev_src, iwSrc, ihSrc,  iwDst,  ihDst);
	}
	checkCudaErrors(hipMemcpy(dst, dev_result, iwDst*ihDst*sizeof(unsigned short), hipMemcpyDeviceToHost));


	checkCudaErrors(hipFree(dev_src));
	checkCudaErrors(hipFree(dev_result));
}

extern "C" void ResizeImage8u(unsigned char *dst, unsigned char *src, int iwSrc, int ihSrc, int iwDst, int ihDst)
{
	unsigned char *dev_result=NULL;
	unsigned char *dev_src=NULL;

	
	checkCudaErrors(hipMalloc(&dev_src, sizeof(unsigned char)*iwSrc*ihSrc));
    checkCudaErrors(hipMemcpy(dev_src, src,sizeof(unsigned char)*iwSrc*ihSrc, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc( &dev_result, iwDst*ihDst*sizeof(unsigned char)));

	dim3 blocks((iwDst+31)/32,(ihDst+15)/16);
	dim3 threads(32,16);

	if(iwDst<iwSrc)
	{
		ResizeDown8u_kernel<<<blocks, threads>>>(dev_result, dev_src, iwSrc, ihSrc,  iwDst,  ihDst);
	}
	else
	{
		ResizeUp8u_kernel<<<blocks, threads>>>(dev_result, dev_src, iwSrc, ihSrc,  iwDst,  ihDst);
	}
	checkCudaErrors(hipMemcpy(dst, dev_result, iwDst*ihDst*sizeof(unsigned char), hipMemcpyDeviceToHost));


	checkCudaErrors(hipFree(dev_src));
	checkCudaErrors(hipFree(dev_result));
}





extern "C" void Zoom(unsigned short *dst, unsigned short *src, int iwDst, int ihDst, float fZoom, float fx, float fy)
{
	unsigned short *dev_result=NULL;
	unsigned short *dev_src=NULL;

	hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned short>();
	
	checkCudaErrors(hipMalloc(&dev_src, sizeof(unsigned short)*iwDst*ihDst));
    checkCudaErrors(hipMemcpy(dev_src, src, sizeof(unsigned short)*iwDst*ihDst, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc( (void **)&dev_result, iwDst*ihDst*sizeof(unsigned short)));


	dim3 blocks((iwDst+31)/32,(ihDst+15)/16);
	dim3 threads(32,16);

	Zoom_kernel<<<blocks, threads>>>(dev_result, dev_src, iwDst,  ihDst, fZoom, fx, fy);

	checkCudaErrors(hipMemcpy(dst, dev_result, iwDst*ihDst*sizeof(unsigned short), hipMemcpyDeviceToHost));


	checkCudaErrors(hipFree(dev_src));
	checkCudaErrors(hipFree(dev_result));
}

extern "C" void ShiftAddImage_dev(float *devDest, int fWidth, int fHeight, unsigned short *dev_src, int iwDst, int ihDst, const float dx, const float dy)
{

	dim3 blocks((iwDst + 31) / 32, (ihDst + 15) / 16);
	dim3 threads(32, 16);


	ShiftAddImage_kernel << <blocks, threads >> >(devDest, fWidth, fHeight, dev_src, iwDst, ihDst, dx, dy);


}

extern "C" void Div32Image_dev(unsigned short *dev_dst, int iwDst, int ihDst, float *dev_src, int fWidth, int fHeight,  float DivC)
{

	dim3 blocks((iwDst + 31) / 32, (ihDst + 15) / 16);
	dim3 threads(32, 16);


	Div32Image_kernel << <blocks, threads >> >(dev_dst, iwDst, ihDst, dev_src, fWidth, fHeight, DivC);


}

extern "C" void CudaMemCopyRect_dev(unsigned short *dev_result, unsigned short * dev_src, int iwSrc, int ihSrc, int iwDst, int ihDst, int sx, int sy)
{
	dim3 blocks((iwDst + 31) / 32, (ihDst + 15) / 16);
	dim3 threads(32, 16);

	CudaMemCopyRect_kernel << <blocks, threads >> > (dev_result, dev_src, iwSrc, ihSrc, iwDst, ihDst, sx, sy);

}



extern "C" void ShiftImage_dev(unsigned short *dev_src, int iwDst, int ihDst, const float dx, const float dy)
{
	unsigned short *dev_result = NULL;
	checkCudaErrors(hipMalloc((void **)&dev_result, iwDst*ihDst * sizeof(unsigned short)));

	dim3 blocks((iwDst + 31) / 32, (ihDst + 15) / 16);
	dim3 threads(32, 16);


	ShiftImage_kernel << <blocks, threads >> >(dev_result, dev_src, iwDst, ihDst, dx, dy);

	checkCudaErrors(hipFree(dev_result));
}

extern "C" void ShiftImage(unsigned short *SrcDst, int iwDst, int ihDst, const float dx, const float dy)
{
	unsigned short *dev_result = NULL;
	unsigned short *dev_src = NULL;


	checkCudaErrors(hipMalloc(&dev_src, sizeof(unsigned short)*iwDst*ihDst));
	checkCudaErrors(hipMemcpy(dev_src, SrcDst, sizeof(unsigned short)*iwDst*ihDst, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void **)&dev_result, iwDst*ihDst * sizeof(unsigned short)));

	dim3 blocks((iwDst + 31) / 32, (ihDst + 15) / 16);
	dim3 threads(32, 16);


	ShiftImage_kernel << <blocks, threads >> >(dev_result, dev_src, iwDst, ihDst, dx, dy);


	checkCudaErrors(hipMemcpy(SrcDst, dev_result, iwDst*ihDst * sizeof(unsigned short), hipMemcpyDeviceToHost));


	checkCudaErrors(hipFree(dev_src));
	checkCudaErrors(hipFree(dev_result));
}


extern "C" void Rotate(unsigned short *SrcDst, int iwDst, int ihDst, float fAngleDeg)
{
	unsigned short *dev_result=NULL;
	unsigned short *dev_src=NULL;

	
	checkCudaErrors(hipMalloc(&dev_src, sizeof(unsigned short)*iwDst*ihDst));
    checkCudaErrors(hipMemcpy(dev_src, SrcDst, sizeof(unsigned short)*iwDst*ihDst, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc( (void **)&dev_result, iwDst*ihDst*sizeof(unsigned short)));

	Rotate_Dev(dev_result, dev_src, iwDst, ihDst, fAngleDeg);


	checkCudaErrors(hipMemcpy(SrcDst, dev_result, iwDst*ihDst*sizeof(unsigned short), hipMemcpyDeviceToHost));


	checkCudaErrors(hipFree(dev_src));
	checkCudaErrors(hipFree(dev_result));
}


extern "C" void Rotate8u(unsigned char* SrcDst, int iwDst, int ihDst, float fAngleDeg)
{
	unsigned char* dev_result = NULL;
	unsigned char* dev_src = NULL;


	checkCudaErrors(hipMalloc(&dev_src, sizeof(unsigned char) * iwDst * ihDst));
	checkCudaErrors(hipMemcpy(dev_src, SrcDst, sizeof(unsigned char) * iwDst * ihDst, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&dev_result, iwDst * ihDst * sizeof(unsigned char)));

	Rotate8u_Dev(dev_result, dev_src, iwDst, ihDst, fAngleDeg);


	checkCudaErrors(hipMemcpy(SrcDst, dev_result, iwDst * ihDst * sizeof(unsigned char), hipMemcpyDeviceToHost));


	checkCudaErrors(hipFree(dev_src));
	checkCudaErrors(hipFree(dev_result));
}



extern "C" void SetC_Dev(unsigned short *dev_SrcDest,int wDst, int hDst, unsigned short ValC)
{
	dim3 blocks((wDst+31)/32,(hDst+15)/16);
	dim3 threads(32,16);

	Set_kernel<<<blocks, threads>>>(dev_SrcDest, wDst, hDst, ValC);
}


__global__ void
Zoom8uxy_kernel(unsigned char *dest, unsigned char *src, int wDst, int hDst, float fZoomx, float fZoomxy)
{
	int ix = threadIdx.x + blockIdx.x*blockDim.x;
	int iy = threadIdx.y + blockIdx.y*blockDim.y;

	int resultPix = 0;
	int offset = ix + iy*wDst;

	bool bAvailable = false;

	if (ix<wDst && iy<hDst)
	{
		float x = ((float)ix - (float)wDst / 2)*fZoomx;
		float y = ((float)iy - (float)hDst / 2)*fZoomxy;

		x += (float)wDst / 2;
		y += (float)hDst / 2;

		bAvailable = true;

		if (x >= 0.0f && x <= wDst - 1 && y >= 0 && y <= hDst - 1)
		{
			int i = int(x);
			int j = int(y);

			float ki1 = (x - (float)i);
			float ki0 = 1.0f - ki1;

			float kj1 = (y - (float)j);
			float kj0 = 1.0f - kj1;

			float SumPix = 0.0f;
			float SumK = 0.0f;


			SumPix += ki0*kj0*src[i + j*wDst];
			SumK += ki0*kj0;

			if (i + 1 <= wDst - 1)
			{
				SumPix += ki1*kj0*src[(i + 1) + j*wDst];
				SumK += ki1*kj0;
			}
			if (j + 1 <= hDst - 1)
			{
				SumPix += ki0*kj1*src[i + (j + 1)*wDst];
				SumK += ki0*kj1;
			}
			if (i + 1 <= wDst - 1 && j + 1 <= hDst - 1)
			{
				SumPix += ki1*kj1*src[(i + 1) + (j + 1)*wDst];
				SumK += ki1*kj1;
			}
			resultPix = max(0, min(255, int(SumPix / SumK + 0.5)));

		}
		else
		{
			resultPix = 0;
		}

	}

	__syncthreads();

	if (bAvailable)
	{
		dest[offset] = resultPix;
	}
}

extern "C" void Zoom8uxy_Dev(unsigned char *dev_result, unsigned char *dev_src, int iwDst, int ihDst, float fZoomx, float fZoomy)
{
	dim3 blocks((iwDst + 31) / 32, (ihDst + 15) / 16);
	dim3 threads(32, 16);

	Zoom8uxy_kernel << <blocks, threads >> >(dev_result, dev_src, iwDst, ihDst, fZoomx, fZoomy);
}

extern "C" void Rotate8u_Dev(unsigned char *dev_result, unsigned char *dev_src, int iwDst, int ihDst, float fAngleRad)
{
	dim3 blocks((iwDst + 31) / 32, (ihDst + 15) / 16);
	dim3 threads(32, 16);

	float cos_deg = cos(fAngleRad);
	float sin_deg = sin(fAngleRad);

	Rotate8u_kernel << <blocks, threads >> >(dev_result, dev_src, iwDst, ihDst, cos_deg, sin_deg);
}




__global__ void
Zoom8u_kernel(unsigned char *dest, unsigned char *src, int DstSrc8u_step, int DstSrc8u_w, int DstSrc8u_h, float fZoom, float dx, float dy)
{
	int ix = threadIdx.x + blockIdx.x*blockDim.x;
	int iy = threadIdx.y + blockIdx.y*blockDim.y;

	int resultPix = 0;
	int dst_offset = ix + iy*DstSrc8u_step;

	bool bAvailable = false;

	if (ix<DstSrc8u_w && iy<DstSrc8u_h)
	{
		float x = ((float)ix - (float)DstSrc8u_w / 2)*fZoom + dx;
		float y = ((float)iy - (float)DstSrc8u_h / 2)*fZoom + dy;

		x += (float)DstSrc8u_w / 2;
		y += (float)DstSrc8u_h / 2;

		bAvailable = true;

		int i = int(x);
		int j = int(y);

		if (i >= 0 && i < DstSrc8u_w && y >= 0 && y < DstSrc8u_h)
		{
			float ki1 = (x - (float)i);
			float ki0 = 1.0f - ki1;

			float kj1 = (y - (float)j);
			float kj0 = 1.0f - kj1;

			float SumPix = 0.0f;
			float SumK = 0.0f;


			SumPix += ki0*kj0*src[i + j*DstSrc8u_step];
			SumK += ki0*kj0;

			if (i + 1 <= DstSrc8u_w - 1)
			{
				SumPix += ki1*kj0*src[(i + 1) + j*DstSrc8u_step];
				SumK += ki1*kj0;
			}
			if (j + 1 <= DstSrc8u_h - 1)
			{
				SumPix += ki0*kj1*src[i + (j + 1)*DstSrc8u_step];
				SumK += ki0*kj1;
			}
			if (i + 1 <= DstSrc8u_w - 1 && j + 1 <= DstSrc8u_h - 1)
			{
				SumPix += ki1*kj1*src[(i + 1) + (j + 1)*DstSrc8u_step];
				SumK += ki1*kj1;
			}
			int result = SumK == 0.0f ? 0 : int(SumPix / SumK + 0.5);
			resultPix = max(0, min(255, result));
		}
		else
		{
			resultPix = src[ix + iy*DstSrc8u_step];
		}

	}

	__syncthreads();

	if (bAvailable)
	{
		dest[dst_offset] = resultPix;
	}
}

extern "C" void Zoom8u_Dev(unsigned char *dev_result, unsigned char *dev_src, int dev_DstSrc_step, int dev_DstSrc_w, int dev_DstSrc_h, float fZoom, float fx, float fy)
{
	dim3 blocks((dev_DstSrc_w + 31) / 32, (dev_DstSrc_h + 15) / 16);
	dim3 threads(32, 16);

	Zoom8u_kernel << <blocks, threads >> >(dev_result, dev_src, dev_DstSrc_step, dev_DstSrc_w, dev_DstSrc_h, fZoom, fx, fy);
}

extern "C" void Zoom8(unsigned char *dst, unsigned char *src, int step, int iwDst, int ihDst, float fZoom)
{
	unsigned char* dev_result = NULL;
	unsigned char* dev_src = NULL;

	hipMalloc(&dev_src, sizeof(unsigned char)*step*ihDst);
	hipMemcpy(dev_src, src, sizeof(unsigned char)*step*ihDst, hipMemcpyHostToDevice);

	hipMalloc(&dev_result, sizeof(unsigned char)*step*ihDst);

	Zoom8u_Dev(dev_result, dev_src, step, iwDst, ihDst, fZoom);

	hipMemcpy(dst, dev_result, sizeof(unsigned char)*step*ihDst, hipMemcpyDeviceToHost);

	hipFree(dev_src);
	hipFree(dev_result);
}